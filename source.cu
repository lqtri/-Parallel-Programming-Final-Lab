
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <limits.h>

__constant__ float dc_xSobel[9];
__constant__ float dc_ySobel[9];

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, 
		int &numChannels, int &width, int &height, uint8_t * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	if (strcmp(type, "P2") == 0)
		numChannels = 1;
	else if (strcmp(type, "P3") == 0)
		numChannels = 3;
	else // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);

	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uint8_t *)malloc(width * height * numChannels);
	for (int i = 0; i < width * height * numChannels; i++)
		fscanf(f, "%hhu", &pixels[i]);

	fclose(f);
}

void writePnm(uint8_t * pixels, int numChannels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printMatrix (uint8_t *in, int width, int height){
	for (int i=0; i<height; i++){
		for (int j=0; j<width; j++){
			printf("%d ", in[i*width+j]);
		}
		printf("\n");
	}
}

__global__ void convertRgb2GrayKernel(uint8_t *inPixels, int width, int height,
                                      uint8_t *outPixels) {
  // TODO
  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int c = blockIdx.x * blockDim.x + threadIdx.x;
  // Reminder: gray = 0.299*red + 0.587*green + 0.114*blue
  if (r < height && c < width) {
    int i = r * width + c;
    outPixels[i] = 0.299 * inPixels[3 * i] + 0.587 * inPixels[3 * i + 1] +
                   0.114 * inPixels[3 * i + 2];
  }
}

__global__ void energyCalculatorKernel(uint8_t *inPixels, int width, int height,
                               	float *xSobel, float* ySobel,
                               	uint8_t *outPixels) {

  int r = blockIdx.y * blockDim.y + threadIdx.y;
  int c = blockIdx.x * blockDim.x + threadIdx.x;

  if (r < height && c < width) {
    int i = r * width + c;
	float xvalue = 0, yvalue = 0;
    for (int r_sobel = 0; r_sobel < 3; r_sobel++) {
      for (int c_sobel = 0; c_sobel < 3; c_sobel++) {

        int i_sobel = r_sobel * 3 + c_sobel;
		int r_img = r - 1 + r_sobel;
		int c_img = c - 1 + c_sobel;
		r_img = min(max(0,r_img), height-1);
		c_img = min(max(0,c_img), width-1);

        int i_img = r_img * width + c_img;
        xvalue += xSobel[i_sobel] * inPixels[i_img];
		yvalue += ySobel[i_sobel] * inPixels[i_img];
      } 
    }
	outPixels[i] = abs(xvalue) + abs(yvalue);
  }
}

__global__ void seamsImportanceCalculator (uint8_t* map, int8_t* backtrack, int width, int height) {
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if (r<height && c<width) {
		int i = r*width+c;
		int j = i - width;

		if (j<0) {
			backtrack[i] = 0;
			return;
		}

		if (c == 0) {
			if (map[j] < map[j+1]){
				map[i] += map[j];
				backtrack[i] = 0;
			}
			else {
				map[i] += map[j+1];
				backtrack[i] = 1;
			}
		}
		else if (c == width-1){
			if (map[j-1] < map[j]){
				map[i] += map[j-1];
				backtrack[i] = -1;
			}
			else {
				map[i] += map[j];
				backtrack[i] = 0;
			}
		}
		else {
			int min = map[j], idx = 0;
			if (map[j-1]<min){
				min = map[j-1];
				idx = -1;
			}
			if (map[j+1]<min){
				min = map[j+1];
				idx = 1;
			}
			map[i] = min;
			backtrack[i] = idx;
		}
	}
}

__global__ void transferDataKernel (uint8_t *dst, uint8_t *src, int width, int height){
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;

	if (r<height && c<width) {	
		int i = r*width+c;
		dst[i] = src[i];
	}
}

__global__ void copyARowKernel (uint8_t *dst, uint8_t *src, int head, int tail){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i+head < tail)
		dst[i] = src[i+head];
}

__global__ void findMinsKernel(uint8_t * in, int n, uint8_t* mins, uint8_t* min_indices)
{
	__shared__ uint8_t sm_min_index[1];
    int numElemsBeforeBlk = blockIdx.x * blockDim.x * 2;
    int i = numElemsBeforeBlk + threadIdx.x;
    for (int stride = blockDim.x; stride >= 1; stride /= 2) {
        if (threadIdx.x < stride){
			if (in[i] > in[i+stride]){
				in[i] = in[i+stride];
				sm_min_index[0] = i+stride;
			}
			else sm_min_index[0] = i;
		}
        __syncthreads(); // Synchronize within each block
    }

    if (threadIdx.x == 0){
        mins[blockIdx.x] = in[numElemsBeforeBlk];
		min_indices[blockIdx.x] = sm_min_index[0];
	}
}

uint8_t findMinIndexFromHost (uint8_t * mins, uint8_t * indices, int n){
	uint8_t min = mins[0];
	uint8_t min_index = indices[0];

	for (int i=1; i<n; i++){
		if (mins[i] < min){
			min = mins[i];
			min_index = indices[i];
		}
	}
	printf("Min energy: %d\nIts index: %d\n", min, min_index);
	return min_index;
}

// __global__ void removeSeamKernel (uint8_t* energy, uint8_t* inPixels, int width, int height, uint8_t* index, int8_t* backtrack){
// }

void seamCarving(uint8_t * inPixels, int width, int height, int new_width, float * xSobel, float * ySobel,
				uint8_t * grayPixels, uint8_t * energy, bool useDevice=false, dim3 blockSize=dim3(1))
{
	GpuTimer timer;
	timer.Start();
	if (useDevice == false)
	{
		// Host running
	}
	else // use device
	{
		size_t nBytes = width * height * sizeof(uint8_t);

		// Host allocates memories on device
		uint8_t *d_inPixels, *d_grayPixels, *d_energy;
		float *d_xSobel, *d_ySobel;
		CHECK(hipMalloc(&d_inPixels, nBytes*3));
		CHECK(hipMalloc(&d_grayPixels, nBytes));
		CHECK(hipMalloc(&d_energy, nBytes));
		CHECK(hipMalloc(&d_xSobel, 9*sizeof(float)));
		CHECK(hipMalloc(&d_ySobel, 9*sizeof(float)));

		// Host copies data to device memories
		CHECK(hipMemcpy(d_inPixels, inPixels, nBytes*3, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_xSobel, xSobel, 9*sizeof(float), hipMemcpyHostToDevice))
		CHECK(hipMemcpy(d_ySobel, ySobel, 9*sizeof(float), hipMemcpyHostToDevice))
		
		// Host invokes kernel function to add vectors on device
		dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
		convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
		hipDeviceSynchronize();
        CHECK(hipGetLastError());

		// Host copies result from device memory
		CHECK(hipMemcpy(grayPixels, d_grayPixels, nBytes, hipMemcpyDeviceToHost));
		
		energyCalculatorKernel<<<gridSize, blockSize>>>(d_grayPixels, width, height, d_xSobel, d_ySobel, d_energy);
		hipDeviceSynchronize();
        CHECK(hipGetLastError());

		// Host copies result from device memory
		CHECK(hipMemcpy(energy, d_energy, nBytes, hipMemcpyDeviceToHost));

		uint8_t *map;
		int8_t *backtrack;
		CHECK(hipMalloc(&map, nBytes));
		CHECK(hipMalloc(&backtrack, nBytes));

		for (int w = width; w > new_width; w--){
			//initialize map (copy from energy), backtrack
			// transferDataKernel<<<gridSize, blockSize>>>(map, d_energy, w, height);
			// cudaDeviceSynchronize();
			// CHECK(cudaGetLastError());
			CHECK(hipMemcpy(map, energy, nBytes, hipMemcpyHostToDevice));

			//Test map correctness
			// uint8_t* testmap = (uint8_t*)malloc(w*height*sizeof(uint8_t));
			// CHECK(cudaMemcpy(testmap, map, nBytes, cudaMemcpyDeviceToHost));
			// printf("\nMAP\n");
			// printMatrix(testmap, w, height);
			// free(testmap);

			//calculate seam importance
			seamsImportanceCalculator<<<gridSize, blockSize>>>(map, backtrack, w, height);
			hipDeviceSynchronize();
			CHECK(hipGetLastError());

			//Test importance
			uint8_t* testimportance = (uint8_t*)malloc(w*height*sizeof(uint8_t));
			CHECK(hipMemcpy(testimportance, map, nBytes, hipMemcpyDeviceToHost));
			printf("\nMAP\n");
			printMatrix(testimportance, w, height);
			free(testimportance);


			dim3 gridSize1((w-1)/(blockSize.x)+1, 1);
			dim3 gridSize2((w-1)/(2*blockSize.x)+1, 1);

			uint8_t* d_last_row;
			CHECK(hipMalloc(&d_last_row, w*sizeof(uint8_t)));
			copyARowKernel<<<gridSize1, blockSize>>>(d_last_row, map, (height-1)*width, height*width);
			hipDeviceSynchronize();
			CHECK(hipGetLastError());

			uint8_t* row = (uint8_t*)malloc(w*sizeof(uint8_t));
			CHECK(hipMemcpy(row, d_last_row, w*sizeof(uint8_t), hipMemcpyDeviceToHost));
			printf("\nLAST SEAM IMPORTANCE ROW\n");
			for (int t = 0; t<w; t++){
				printf("%d ", row[t]);
			}
			printf("\n");
			free(row);

			size_t mins_size = gridSize2.x * sizeof(uint8_t);
			uint8_t* mins = (uint8_t*) malloc(mins_size);
			uint8_t* min_indices = (uint8_t*) malloc(mins_size);		

			uint8_t* d_mins;
			uint8_t* d_min_indices;
			CHECK(hipMalloc(&d_mins, mins_size));
			CHECK(hipMalloc(&d_min_indices, mins_size));

			findMinsKernel<<<gridSize2, blockSize>>>(d_last_row, w, d_mins, d_min_indices);
			hipDeviceSynchronize();
			CHECK(hipGetLastError());

			CHECK(hipMemcpy(mins, d_mins, mins_size, hipMemcpyDeviceToHost));
			CHECK(hipMemcpy(min_indices, d_min_indices, mins_size, hipMemcpyDeviceToHost));

			printf("\nMIN ELEMENT FROM DEVICE\n");
			for (int t = 0; t < gridSize2.x; t++){
				printf("%d ", mins[t]);
			}
			printf("\n");
			for (int t = 0; t < gridSize2.x; t++){
				printf("%d ", min_indices[t]);
			}
			printf("\n");

			printf("\nFIND MIN ENERGY\n");
			findMinIndexFromHost(mins, min_indices, gridSize2.x);

			free(mins);
			free(min_indices);
			CHECK(hipFree(d_mins));
			CHECK(hipFree(d_min_indices));
			CHECK(hipFree(d_last_row));
			
			break;
		}

		// Host frees device memories
		CHECK(hipFree(d_inPixels));
		CHECK(hipFree(d_grayPixels));
		CHECK(hipFree(d_energy));
		CHECK(hipFree(d_xSobel));
		CHECK(hipFree(d_ySobel));
		CHECK(hipFree(map));
		CHECK(hipFree(backtrack));
	}
	timer.Stop();
	float time = timer.Elapsed();
	printf("Processing time (%s): %f ms\n\n", useDevice == true? "use device" : "use host", time);
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");
}

int main(int argc, char ** argv)
{
	printDeviceInfo();

	// Read input image file
	int numChannels, width, height;
	uint8_t * inPixels;
	readPnm(argv[1], numChannels, width, height, inPixels);

	int new_width = 450;
	printf("\nImage size (width x height): %i x %i\n", width, height);
	// printf("\nEnter a number of new width size to scale: ");
	// scanf("%d", &new_width);

    float xSobel[] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
	float ySobel[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};

	// Convert RGB to grayscale using device
	uint8_t *grayPixels = (uint8_t *)malloc(width * height);
	uint8_t *energy;
	hipHostMalloc(&energy, width * height * sizeof(uint8_t), hipHostMallocDefault);

	dim3 blockSize(32, 32); // Default
	if (argc == 4) {
		blockSize.x = atoi(argv[2]);
		blockSize.y = atoi(argv[3]);
	}

	seamCarving(inPixels, width, height, new_width, xSobel, ySobel, grayPixels, energy, true, blockSize);
	char *outFileNameBase = strtok(argv[1], "."); // Get rid of extension
	writePnm(grayPixels,1, width, height, concatStr(outFileNameBase, "_grayscale.pnm"));
	writePnm(energy,1, width, height, concatStr(outFileNameBase, "_energy.pnm"));

	// Free memories
	free(inPixels);
	free(grayPixels);
	hipHostFree(energy);

	return 0;
}